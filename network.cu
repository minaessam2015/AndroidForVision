#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include<hipDNN.h>
#include <stdio.h>
#include "opencv2/core/core.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include<hipblas.h>
#include<string>
#include<fstream>
#include<cmath>
#include<ctime>

#define TEST 1
using namespace std;


__global__ void flatten(float* input, float* out, int batch, int n, int h, int w) {
	int cond1 = (((threadIdx.y*w) + threadIdx.x));
	int cond2 = (threadIdx.x*n) + blockIdx.x + threadIdx.y*w*n;
	if (cond2<(n*h*w)) {
		printf("index  %d    %d    \n", ((threadIdx.y*w) + threadIdx.x) + (blockIdx.x*w*h), cond2);
		//printf("%f\n", input[((threadIdx.y*w) + threadIdx.x) ]);
		out[cond1 + (blockIdx.x*w*h)] = input[cond2];
	}
}

__global__ void addBias(float* vector,float bias,int size) {
	int index = blockIdx.y*blockIdx.x + blockIdx.x;
	if (index < size) {
		//printf(" from addBias  %f \n", vector[index]);
		vector[index] += bias;
		//printf(" from addBias  %f \n", vector[index]);
	}
}

void readWeights(float weights[][3][3][3], int m/*output*/, int n/*input*/, int h, int w, string baseFileName) {

	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			string fileName = "weights/"+baseFileName + std::to_string(j) + "X" + std::to_string(i) + ".txt";
			ifstream in(fileName, std::ifstream::in);
			//cout << fileName << "\n";

			char c;
			if (!in.is_open())
			
			{
				cout << "file didn't open \n";
				return;
			}
			string s = "";
			for (int k = 0; k < h; k++) {
				//cout << s.length()<<"\n";
				for (int l = 0; l < w; l++) {
					//cout << "L " << l;
					while (in.get(c)) {
						if (c == ' '&&s.length() == 0)continue;
						if (c != ' '&&c != '\n')s += c;
						else
						{
							break;
						}
						//cout << c << " ";
					}
					if (s.length()>0)weights[i][j][k][l] = std::stof(s);
					s = "";
					
				}
			}
		}
	}

}

void readWeights(float* weights, int size, string baseFileName) {
	ifstream in("weights/" + baseFileName, std::ifstream::in);
	//cout << baseFileName << "\n";
	if (!in.is_open())
	
	{
		cout << "file didn't open \n";
		return;
	}
	char c;

	string s = "";
	for (int i = 0; i < size; i++) {


		while (in.get(c)) {
			if (c == ' '&&s.length() == 0)continue;
			if (c != ' '&&c != '\n')s += c;
			else
			{
				break;
			}

		}
		if (s.length()>0) weights[i] = std::stof(s);
		//cout << i<<"  "<<s << "\n";
		s = "";
	}
	in.close();

}



void readWeights(float weights[][8][3][3], int m/*output*/, int n/*input*/, int h, int w, string baseFileName) {
	//file will be in format baseFileName nXm .txt
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			string fileName = "weights/" + baseFileName + std::to_string(j) + "X" + std::to_string(i) + ".txt";
			ifstream in(fileName, std::ifstream::in);
			//cout << fileName << "\n";

			char c;
			if (!in.is_open())
			
			{
				cout << "file didn't open \n";
				return;
			}
			string s = "";
			for (int k = 0; k < h; k++) {
				//cout << s.length()<<"\n";
				for (int l = 0; l < w; l++) {
					//cout << "L " << l;
					while (in.get(c)) {
						if (c == ' '&&s.length() == 0)continue;
						if (c != ' '&&c != '\n')s += c;
						else
						{

							//cout << "breaking with c " << c << "\n";
							break;
						}
						//cout << c << " ";
					}
					if (s.length()>0)weights[i][j][k][l] = std::stof(s);
					s = "";
					//cout << std::stof(s) << "\n";
				}
			}
		}
	}

}


cv::Mat load_image(const char* image_path) {
	cv::Mat image = cv::imread(image_path, CV_LOAD_IMAGE_COLOR);
	if (image.empty()) { cerr << "couldn't open image\n"; }
	cv::cvtColor(image, image, cv::COLOR_BGR2RGB);
	image.convertTo(image, CV_32FC3);
	cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
	return image;
}

void save_image(const char* output_filename,
	float* buffer,
	int height,
	int width) {
	cv::Mat output_image(height, width, CV_32FC3, buffer);
	// Make negative values zero.
	cv::threshold(output_image,
		output_image,
		/*threshold=*/0,
		/*maxval=*/0,
		cv::THRESH_TOZERO);
	cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
	output_image.convertTo(output_image, CV_8UC3);
	cv::imwrite(output_filename, output_image);
}

#define cudnnCheck(exp)																\
{																					\
hipdnnStatus_t status=(exp);															\
if(status!=HIPDNN_STATUS_SUCCESS){													\
cerr<<"Error at line "<<__LINE__<<hipdnnGetErrorString(status)<<"\n";				\
std::exit(EXIT_FAILURE);															\
}																					\
																					\
} 

int main() {


	float* alpha=new float;
	alpha[0] = 1.0;
	float* beta=new float;
	beta[0] = 0.0;


	char* imageName = "car2.png";
	cout << imageName << "\n";
	cv::Mat image = load_image(imageName);
	//for (int i = 0; i < 5; i++) {
	//	cout<<(image.at<float>(i, 0))<<" ";
	//}
	//cout << "\n";
	hipdnnHandle_t cudnn;
	cudnnCheck(hipdnnCreate(&cudnn));
	cout << "image dims " << image.rows << " X " << image.cols << "\n";
	//input image
	long long t1 = clock();
	hipdnnTensorDescriptor_t inputImageDes;
	cudnnCheck(hipdnnCreateTensorDescriptor(&inputImageDes));
	cudnnCheck(hipdnnSetTensor4dDescriptor(inputImageDes,
		HIPDNN_TENSOR_NHWC,
		HIPDNN_DATA_FLOAT,
		1,
		3,
		32,
		32
	));
//-----------------------------------------------------------------CONV1------------------------------------------------------------
	//W1
	hipdnnFilterDescriptor_t conv1W;
	cudnnCheck(hipdnnCreateFilterDescriptor(&conv1W));
	cudnnCheck(hipdnnSetFilter4dDescriptor(conv1W,
		HIPDNN_DATA_FLOAT,
		HIPDNN_TENSOR_NCHW,
		8,
		3,
		3,
		3));
	hipdnnTensorDescriptor_t conv1Out;
	cudnnCheck(hipdnnCreateTensorDescriptor(&conv1Out));
	cudnnCheck(hipdnnSetTensor4dDescriptor(conv1Out,
		HIPDNN_TENSOR_NHWC,
		HIPDNN_DATA_FLOAT,
		1,
		8,
		32,
		32));

	hipdnnConvolutionDescriptor_t conv1Des;
	cudnnCheck(hipdnnCreateConvolutionDescriptor(&conv1Des));
	cudnnCheck(hipdnnSetConvolution2dDescriptor(conv1Des,
		1,
		1,
		1,
		1,
		1,
		1,
		HIPDNN_CROSS_CORRELATION,
		HIPDNN_DATA_FLOAT));

	hipdnnConvolutionFwdAlgo_t conv1AlgDes;
	cudnnCheck(hipdnnGetConvolutionForwardAlgorithm(cudnn,
		inputImageDes,
		conv1W,
		conv1Des,
		conv1Out,
		HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
		0,
		&conv1AlgDes));

	size_t workspace_bytes = 0;


	
	cudnnCheck(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
		inputImageDes,
		conv1W,
		conv1Des,
		conv1Out,
		conv1AlgDes,
		&workspace_bytes));
	//cout << "required space for conv1 " << workspace_bytes / (1024) << "\n";
	
	void* d_workspace = nullptr;
	hipMalloc(&d_workspace, workspace_bytes);
	//image alloc 
	float* d_image;
	int imageSize = 3 * image.rows*image.cols;
	hipMalloc(&d_image, imageSize * sizeof(float));
	hipMemcpy(d_image, image.ptr<float>(0), imageSize*sizeof(float), hipMemcpyHostToDevice);



	//output from conv1 [ 1 * 8 * 32 * 32 ]
	float* d_conv1Out;
	int conv1OutSize = 1 * 8 * 32 * 32;
	hipMalloc(&d_conv1Out, conv1OutSize * sizeof(float));
	
	//conv1 kernel [3 * 3 * 3 * 8]
	float* d_conv1W;
	int conv1WSize = 3 * 3 * 3 * 8;
	hipMalloc(&d_conv1W, conv1WSize * sizeof(float));
	//TODO
	//get the data 
	//copy the data to the GPU
	float h_conv1W[8][3][3][3];
	readWeights(h_conv1W, 8, 3, 3, 3,"conv1Weights");
	hipMemcpy(d_conv1W, h_conv1W, sizeof(h_conv1W), hipMemcpyHostToDevice);
	//cout << "conv1 weights\n";
	//
	////test for conv1 weights
	//float h_conv1WTest[20];
	//hipMemcpy(h_conv1WTest, d_conv1W, sizeof(h_conv1WTest), hipMemcpyDeviceToHost);
	//for (int i = 0; i < 20; i++) {
	//	cout << h_conv1WTest[i] << " ";
	//}
	//cout << "\n";
	cudnnCheck(hipdnnConvolutionForward(cudnn,
		alpha,
		inputImageDes,
		d_image,
		conv1W,
		d_conv1W,
		conv1Des,
		conv1AlgDes,
		d_workspace,
		workspace_bytes,
		beta,
		conv1Out,
		d_conv1Out));

	////test relu 1 out 
	//float h_conv1Test[20];
	//hipMemcpy(h_conv1Test, d_conv1Out, sizeof(h_conv1Test), hipMemcpyDeviceToHost);
	//cout << "conv1 20 values\n";
	//for (int i = 0; i < 20; i++) {
	//	cout << h_conv1Test[i] << " ";
	//}
	//cout << "\n\n";

//------------------------------------------------------------RELU 1--------------------------------------------------------

	//relu1 in=[1 *32 *32 *8] out the same
	hipdnnActivationDescriptor_t relu1Des;
	cudnnCheck(hipdnnCreateActivationDescriptor(&relu1Des));
	cudnnCheck(hipdnnSetActivationDescriptor(relu1Des,
		HIPDNN_ACTIVATION_RELU,
		HIPDNN_NOT_PROPAGATE_NAN,
		0.0));

	//// INPUT data is same from conv 1 
	////allocate output data [1 *32 *32 *8] 


	//hipdnnTensorDescriptor_t relu1InputDes;
	//cudnnCheck(hipdnnCreateTensorDescriptor(&relu1InputDes));
	//cudnnCheck(hipdnnSetTensor4dDescriptor(relu1InputDes,
	//	HIPDNN_TENSOR_NHWC,
	//	HIPDNN_DATA_FLOAT,
	//	1,
	//	8,
	//	32,
	//	32));

	hipdnnTensorDescriptor_t relu1OutDes;
	cudnnCheck(hipdnnCreateTensorDescriptor(&relu1OutDes));
	cudnnCheck(hipdnnSetTensor4dDescriptor(relu1OutDes,
		HIPDNN_TENSOR_NHWC,
		HIPDNN_DATA_FLOAT,
		1,
		8,
		32,
		32));
	float* d_relu1Out;
	int relu1Size = 1 * 32 * 32 * 8;
	hipMalloc(&d_relu1Out, relu1Size * sizeof(float));


	
	cudnnCheck(hipdnnActivationForward(cudnn,
		relu1Des,
		alpha,
		conv1Out,
		d_conv1Out,
		beta,
		relu1OutDes,
		d_relu1Out
		));
	
	////test relu 1 out 
	//float h_relu1Test[20];
	//hipMemcpy(h_relu1Test, d_relu1Out, sizeof(h_relu1Test), hipMemcpyDeviceToHost);
	//cout << "relu1 20 values\n";
	//for (int i = 0; i < 20; i++) {
	//	cout << h_relu1Test[i] << " ";
	//}
	//cout << "\n\n";
	//----------------------------------------MAX 1 pooling ---------------------------------
	//MAX polling layer 
	//in d_relu1Out
	//out [1 * 4 * 4 * 8]

	hipdnnPoolingDescriptor_t max1Des;
	cudnnCheck(hipdnnCreatePoolingDescriptor(&max1Des));
	cudnnCheck(hipdnnSetPooling2dDescriptor(max1Des,
		HIPDNN_POOLING_MAX,
		HIPDNN_NOT_PROPAGATE_NAN,
		8,
		8,
		0,
		0,
		8,
		8));

	hipdnnTensorDescriptor_t max1OutputDes;
	cudnnCheck(hipdnnCreateTensorDescriptor(&max1OutputDes));
	cudnnCheck(hipdnnSetTensor4dDescriptor(max1OutputDes,
		HIPDNN_TENSOR_NHWC,
		HIPDNN_DATA_FLOAT,
		1,
		8,
		4,
		4));

	float* d_max1Out;
	int max1Size = 1 * 4 * 4 * 8;
	hipMalloc(&d_max1Out, max1Size* sizeof(float));
	
	cudnnCheck(hipdnnPoolingForward(cudnn,
		max1Des,
		alpha,
		relu1OutDes,
		d_relu1Out,
		beta,
		max1OutputDes,
		d_max1Out
		));

	////Test
	//float h_max1Test[20];
	//hipMemcpy(h_max1Test, d_max1Out, sizeof(h_max1Test), hipMemcpyDeviceToHost);
	//cout << "max1 20 values\n";
	//for (int i = 0; i < 20; i++) {
	//	cout << h_max1Test[i] << " ";
	//}
	//cout << "\n\n";

	//-------------------------------------------conv2 layer---------------------------------------------

	//hipdnnTensorDescriptor_t conv2InputDes;
	//cudnnCheck(hipdnnCreateTensorDescriptor(&conv2InputDes));
	//cudnnCheck(hipdnnSetTensor4dDescriptor(conv2InputDes,
	//	HIPDNN_TENSOR_NHWC,
	//	HIPDNN_DATA_FLOAT,
	//	1,
	//	8,
	//	4,
	//	4
	//));
	//W1
	hipdnnFilterDescriptor_t conv2W;
	cudnnCheck(hipdnnCreateFilterDescriptor(&conv2W));
	cudnnCheck(hipdnnSetFilter4dDescriptor(conv2W,
		HIPDNN_DATA_FLOAT,
		HIPDNN_TENSOR_NCHW,
		16,
		8,
		3,
		3));
	hipdnnTensorDescriptor_t conv2Out;
	cudnnCheck(hipdnnCreateTensorDescriptor(&conv2Out));
	cudnnCheck(hipdnnSetTensor4dDescriptor(conv2Out,
		HIPDNN_TENSOR_NHWC,
		HIPDNN_DATA_FLOAT,
		1,
		16,
		4,
		4));

	hipdnnConvolutionDescriptor_t conv2Des;
	cudnnCheck(hipdnnCreateConvolutionDescriptor(&conv2Des));
	cudnnCheck(hipdnnSetConvolution2dDescriptor(conv2Des,
		1,
		1,
		1,
		1,
		1,
		1,
		HIPDNN_CROSS_CORRELATION,
		HIPDNN_DATA_FLOAT));
	hipdnnConvolutionFwdAlgo_t conv2AlgDes;
	cudnnCheck(hipdnnGetConvolutionForwardAlgorithm(cudnn,
		max1OutputDes,
		conv2W,
		conv2Des,
		conv2Out,
		HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
		0,
		&conv2AlgDes));

	size_t workspace_bytes2 = 0;
	cudnnCheck(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
		max1OutputDes,
		conv2W,
		conv2Des,
		conv2Out,
		conv2AlgDes,
		&workspace_bytes2));
	//cout << "space for conv2 " << workspace_bytes2 / 1024 << "\n";
	void* d_workspace2 = nullptr;
	hipMalloc(&d_workspace2, workspace_bytes2);


	//output from conv2 [ 1 * 4 * 4 * 16 ]
	float* d_conv2Out;
	int conv2OutSize = 1 * 4 * 4 * 16;
	hipMalloc(&d_conv2Out, conv2OutSize*sizeof(float));
	

	//conv2 kernel [3 * 3 * 8 * 16]
	float* d_conv2W;
	int conv2WSize = 3 * 3 * 8 * 16;
	hipMalloc(&d_conv2W, conv2WSize*sizeof(float));
	//TODO
	//get the data W2
	//copy the data to the GPU

	float h_conv2W[16][8][3][3];
	readWeights(h_conv2W, 16, 8, 3, 3, "conv2Weights");
	hipMemcpy(d_conv2W, h_conv2W, sizeof(h_conv2W), hipMemcpyHostToDevice);

	cudnnCheck(hipdnnConvolutionForward(cudnn,
		alpha,
		max1OutputDes,
		d_max1Out,
		conv2W,
		d_conv2W,
		conv2Des,
		conv2AlgDes,
		d_workspace2,
		workspace_bytes2,
		beta,
		conv2Out,
		d_conv2Out));

	////Test
	//float h_conv2Test[20];
	//hipMemcpy(h_conv2Test, d_conv2Out, sizeof(h_conv2Test), hipMemcpyDeviceToHost);
	//cout << "conv2 20 values\n";
	//for (int i = 0; i < 20; i++) {
	//	cout << h_conv2Test[i] << " ";
	//}
	//cout << "\n\n";

//-------------------------------------------------------------RELU 2--------------------------------------------------------

	//relu2 in=[1 *4 *4 *8] out the same
	hipdnnActivationDescriptor_t relu2Des;
	cudnnCheck(hipdnnCreateActivationDescriptor(&relu2Des));
	cudnnCheck(hipdnnSetActivationDescriptor(relu2Des,
		HIPDNN_ACTIVATION_RELU,
		HIPDNN_NOT_PROPAGATE_NAN,
		0.0));

	// INPUT data is same from conv 1 
	//allocate output data [ 1 *4 *4 *16 ] 


	//hipdnnTensorDescriptor_t relu1InputDes;
	//cudnnCheck(hipdnnCreateTensorDescriptor(&relu1InputDes));
	//cudnnCheck(hipdnnSetTensor4dDescriptor(relu1InputDes,
	//	HIPDNN_TENSOR_NHWC,
	//	HIPDNN_DATA_FLOAT,
	//	1,
	//	8,
	//	32,
	//	32));

	hipdnnTensorDescriptor_t relu2OutDes;
	cudnnCheck(hipdnnCreateTensorDescriptor(&relu2OutDes));
	cudnnCheck(hipdnnSetTensor4dDescriptor(relu2OutDes,
		HIPDNN_TENSOR_NHWC,
		HIPDNN_DATA_FLOAT,
		1,
		16,
		4,
		4));

	float* d_relu2Out;
	int relu2Size = 1 * 4 * 4 * 16;
	hipMalloc(&d_relu2Out, relu2Size*sizeof(float));


	cudnnCheck(hipdnnActivationForward(cudnn,
		relu2Des,
		alpha,
		conv2Out,
		d_conv2Out,
		beta,
		relu2OutDes,
		d_relu2Out
	));


	////Test
	//float h_relu2Test[4*4*16];
	//hipMemcpy(h_relu2Test, d_relu2Out, sizeof(h_relu2Test), hipMemcpyDeviceToHost);
	//cout << "relu2 16*16 values\n";
	//for (int i = 0; i < 4*4*16; i++) {
	//	cout << h_relu2Test[i] << " ";
	//}
	//cout << "\n\n";

//------------------------------------------------------MAX 2 pooling ----------------------------------------
	//MAX polling layer 
	//in d_relu2Out
	//out [ 1 * 1 * 1 * 16 ]

	hipdnnPoolingDescriptor_t max2Des;
	cudnnCheck(hipdnnCreatePoolingDescriptor(&max2Des));
	cudnnCheck(hipdnnSetPooling2dDescriptor(max2Des,
		HIPDNN_POOLING_MAX,
		HIPDNN_NOT_PROPAGATE_NAN,
		4,
		4,
		0,
		0,
		4,
		4));

	hipdnnTensorDescriptor_t max2OutputDes;
	cudnnCheck(hipdnnCreateTensorDescriptor(&max2OutputDes));
	cudnnCheck(hipdnnSetTensor4dDescriptor(max2OutputDes,
		HIPDNN_TENSOR_NHWC,
		HIPDNN_DATA_FLOAT,
		1,
		16,
		1,
		1));

	float* d_max2Out;
	int max2Size = 1 * 1 * 1 * 16;
	hipMalloc(&d_max2Out, max2Size*sizeof(float));

	cudnnCheck(hipdnnPoolingForward(cudnn,
		max2Des,
		alpha,
		relu2OutDes,
		d_relu2Out,
		beta,
		max2OutputDes,
		d_max2Out
	));

	//float* d_flatOut;
	//hipMalloc(&d_flatOut,max2Size*sizeof(float));

	//hipError_t error;
	//flatten<<<>>>(d_max2Out, d_flatOut, 1, 16, 1, 1);
	//error=hipDeviceSynchronize();
	//if (error != hipSuccess) cout << "error flatten\n";



	////Test
	//float h_max2Test[16];
	//hipMemcpy(h_max2Test, d_max2Out, sizeof(h_max2Test), hipMemcpyDeviceToHost);
	//cout << "max2 16 values\n";
	//for (int i = 0; i <  16; i++) {
	//	cout << h_max2Test[i] << " ";
	//}
	//cout << "\n\n";

//----------------------------------------------------Fully connected-----------------------------------------------------------

	hipblasHandle_t cublas;
	hipblasCreate(&cublas);

	//create the fully connected weights & copy to GPU

	float h_fullyWeights[16];
	readWeights(h_fullyWeights, 16, "fc1Weights.txt");
	float* d_fullyWeights;
	float bias = 0;
	readWeights(&bias, 1, "fc1Bias.txt");
	hipMalloc(&d_fullyWeights, sizeof(h_fullyWeights));
	hipMemcpy(d_fullyWeights, h_fullyWeights, sizeof(h_fullyWeights), hipMemcpyHostToDevice);

	//float h_max2Out[16];
	//hipMemcpy(h_max2Out, d_max2Out, sizeof(h_fullyWeights), hipMemcpyDeviceToHost);
	//cout << "fully connected weights\n";
	//for (int i = 0; i < 16; i++)cout << h_fullyWeights[i] << " ";
	//cout << "\n\n";

	float* d_tmp;
	hipMalloc(&d_tmp,sizeof(float));

	//float fullyBias=0;
	//hipMemcpy(d_tmp, &fullyBias, sizeof(float), hipMemcpyHostToDevice);

	hipblasStatus_t s= hipblasSgemm(cublas,
		HIPBLAS_OP_N, HIPBLAS_OP_N,
		1, 1, 16,
		alpha,
		d_max2Out,  1,
		d_fullyWeights, 16,
		beta,
		d_tmp,  1);
	if (s != HIPBLAS_STATUS_SUCCESS) {
		std::cout << "error cublas\n";
	}
	addBias <<<dim3(1,1), dim3(1) >> > (d_tmp,bias,1);

	hipdnnActivationDescriptor_t activationDes;
	cudnnCheck(hipdnnCreateActivationDescriptor(&activationDes ));
	cudnnCheck(hipdnnSetActivationDescriptor(activationDes,
		HIPDNN_ACTIVATION_SIGMOID,
		HIPDNN_PROPAGATE_NAN,
		0));
	hipdnnTensorDescriptor_t inputToFully;
	cudnnCheck(hipdnnCreateTensorDescriptor(&inputToFully));
	cudnnCheck(hipdnnSetTensor4dDescriptor(inputToFully,
		HIPDNN_TENSOR_NHWC,
		HIPDNN_DATA_FLOAT,
		1,
		1,
		1,
		1));

	cudnnCheck(hipdnnActivationForward(cudnn,
		activationDes,
		alpha,
		inputToFully,
		d_tmp,
		beta,
		inputToFully,
		d_tmp));
	cout << "total time elapsed " <<clock()-t1 << "\n";
	float result=3;
	hipMemcpy(&result, d_tmp, sizeof(float), hipMemcpyDeviceToHost);
	cout << result << "\n";
	
	cudnnCheck(hipdnnDestroyTensorDescriptor(inputImageDes));
	cudnnCheck(hipdnnDestroyTensorDescriptor(conv1Out));
	cudnnCheck(hipdnnDestroyConvolutionDescriptor(conv1Des));
	cudnnCheck(hipdnnDestroyFilterDescriptor(conv1W));
	//continue


}